#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// sweep-tt-multistart.c - using VELOCITYBOX and FLOATBOX
// vim: set tabstop=2 softtabstop=2 shiftwidth=2 expandtab :
////////////////////////////////////////////////////////////////////////////////

/********************************************************************************/
/* Given a velocity field v[nx][ny][nz] for a set of points (i,j,k) (where	*/
/* 0 <= i < nx, 0 <= j < ny, 0 <= k < nz) layed out on a grid with delta unit	*/
/* distance, compute the minimum travel time, tt[nx][ny][nz][numstart], for all	*/
/* points to the numstart starting points.  The program is called as follows:	*/
/*										*/
/*	sweep-tt-multistart vfile fsfile startfile       			*/
/*										*/
// vfile is the velocity field file and has the .vbox format.
/*										*/
/* fsfile is the forward star offset file and has the format:			*/
/*										*/
/*	starsize								*/
/*	oi oj ok for every forward star offset (oi,oj,ok)			*/
/*										*/
/* startfile contains starting points and has the format:			*/
/*										*/
/*	numstart								*/
/*	si sj sk for every starting point					*/
/*										*/
/* The program writes to "output.tt" the following:				*/
/*										*/
/*	nx ny nz								*/
/*	tt[i][j][k] for every point (i,j,k) in row-major order			*/
/*										*/
/* for every starting point.							*/
/* (Note, the program currently exits before this is done.)			*/
/********************************************************************************/

#include "iovelocity.h"
#include "timing.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>

#define	FSRADIUSMAX	7	/* maximum radius forward star */
#define	FSMAX		818	/* maximum # of points in a forward star */
#define MODELMAX	250	/* maximum model dimension in X,Y,Z */
#define STARTMAX	4	/* maximum starting points */

#define GRIDX 256
#define GRIDY 128
#define GRIDZ 1
#define BLOCKX 1
#define BLOCKY 2
#define BLOCKZ 64
const int starSplit[4] = {0, 330, 560, 818};

struct FS {			/* forward start offset */
  int		i, j, k;	/* point coordinates */
  float		d;		/* distance to star center (0,0,0)*/
};

struct MODEL {			/* model point */
  float		v;		/* velocity */
  float		tt[STARTMAX];	/* travel time for starting points */
};

struct START {			/* starting point */
  int		i, j , k;	/* point coordinates */
};

int		changed[STARTMAX];

struct FS	fs[FSMAX];
__constant__ struct FS dc_fs[FSMAX];
struct START	start[STARTMAX];
struct VELOCITYBOX vbox; // stores JUST velocities
struct FLOATBOX ttboxes[STARTMAX]; // stores JUST travel times, one volume per starting point
__device__ int dc_test = 0;

void cudaRun(int, int);
__global__ 
void cudaWorker(
    int d_nx, int d_ny, int d_nz,
    int d_starstart, int d_starend,
    struct FS *pd_fs,
    float *pd_vboxflat,
    float *pd_ttboxflat,
    int *pd_anychange
);
__device__ int 
sweepXYZ(
    int nx, int ny, int nz, 
    int starstart, int starstop,
    struct FS *fs,
    float *vboxflat,
    float *ttboxflat
);
__global__ void 
MergeBoxes(
    float *pd_ttboxflat, 
    float *pd_ttbflatMergeCache, 
    int devNum, 
    int nCells
);

int main(int argc, char* argv[]) {
  int		i, j, k, nx, ny, nz, s;
  int		numradius, starsize, numstart;
  int		fsindex[FSRADIUSMAX];
  float		delta;
  FILE		*fsfile, *ttfile, *startfile;

  const char *velocity_model_file = argv[1];

  /* open velocity model file */
  printf( "Loading velocity model file: %s...", velocity_model_file ); fflush( stdout );
  //if( !vboxloadbinary( &vbox, velocity_model_file ) ) {
  if( !vboxloadtext( &vbox, velocity_model_file ) ) {
    printf( "Cannot open velocity model file: %s\n", velocity_model_file );
    exit(1);
  }
  nx = vbox.box.nx;
  ny = vbox.box.ny;
  nz = vbox.box.nz;
  printf( " done.\n" ); fflush( stdout );
  printf( "Velocity model dimensions: %d x %d x %d\n", nx, ny, nz );

  /* open forward star offset file */
  fsfile = fopen(argv[2],"r");
  if(fsfile == NULL) {
    printf("Cannot open forward star offset file: %s\n", argv[2]);
    exit(1);
  }
  printf("Forward star offset file: %s\n", argv[2]);

  /* open file with starting points */
  startfile = fopen(argv[3],"r");
  if(startfile == NULL) {
    printf("Cannot open starting points file: %s\n", argv[4]);
    exit(1);
  }
  printf("Starting points file: %s\n", argv[3]);

  /* get delta */
  delta = 10.0;
  printf("Delta: %f\n", delta);

  /* read forward star offsets */
  starsize = 0;
  fscanf(fsfile, "%i", &starsize);
  printf("Forward star size: %d\n", starsize);

  for (i=0; i<FSRADIUSMAX; i++) {
    fsindex[i] = 0;
  }
  numradius = 0;
  for (i=0; i<starsize; i++) {
    fscanf(fsfile, "%i %i %i", &fs[i].i, &fs[i].j, &fs[i].k);
    fs[i].d = sqrt(fs[i].i*fs[i].i + fs[i].j*fs[i].j + fs[i].k*fs[i].k);
    if ((numradius+1) < fs[i].d) {
      fsindex[numradius] = i;
      numradius++;
    }
    fs[i].d = delta * fs[i].d; 
  }
  printf("Forward star offsets read\n");
  for (i=0; i<FSRADIUSMAX; i++) {
    printf("numradius: %d, fsindex[%d]: %d\n", numradius, i, fsindex[i]);
  }

  /* read starting points */
  fscanf(startfile, "%i", &numstart);
  // initialize travel times for all starting points
  for( s = 0; s < numstart; s++ ) {
    // prepare travel time volumes
    boxalloc( &ttboxes[s], nx, ny, nz );
    boxsetall( ttboxes[s], INFINITY );

    // set the starting point to have a travel time of 0
    fscanf( startfile, "%i %i %i", &i, &j, &k );
    boxput( ttboxes[s], i, j, k, 0 );
    printf( "starting point %d: %d %d %d\n", s, i, j, k );
    start[s].i = i; start[s].j = j; start[s].k = k;
  }
  printf("Starting points read\n");
  
  int nDevices;
  hipGetDeviceCount(&nDevices);
  printf("device: %d\n", nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
  
	cudaRun(numstart, starsize);

  /* print travel times */
  ttfile = fopen("output.tt","w");
  if(ttfile == NULL) {
    printf("Can not open travel time output file: %s\n", "output.tt");
    exit(1);
  }
  fprintf(ttfile, "%d %d %d\n", nx, ny, nz);
  for (s=0; s<numstart; s++) {
    fprintf(ttfile, "starting point: %d\n", s);
    for (i=0; i<nx; i++) {
      for (j=0; j<ny; j++) {
        for (k=0; k<nz; k++) {
          /* use %g for doubles */

          fprintf(ttfile, "travel time for (%d,%d,%d): %f %d %d %d\n",
            i, j, k, boxget( ttboxes[s], i, j, k ), 0, 0, 0 );
        }
      }
    }
  }
} /* main */

void cudaRun(
    int numstart, 
    int starsize
)
{
  //constants
  const int blkNum = GRIDX * GRIDY * GRIDZ;
  //const int blkSize = BLOCKX * BLOCKY * BLOCKZ;
  //const int tNum = blkNum * blkSize;

  //host variables
  int anychange[DEVNUM][blkNum];
  int i, j, nx = vbox.box.nx, ny = vbox.box.ny, nz = vbox.box.nz, devIdx = 0, devNum = DEVNUM;
  int nCells = nx * ny * nz;
  size_t flatbytes = (size_t)nCells * sizeof(float);
  hipError_t err;
  
  //Cuda variables
  dim3 gridDim(GRIDX,GRIDY,GRIDZ);
  dim3 blockDim(BLOCKX,BLOCKY,BLOCKZ);
  float *ppd_vboxflat[DEVNUM];
  float *pppd_ttboxflat[DEVNUM][STARTMAX];
  float *ppd_ttbflatMergeCache[DEVNUM];
  int *ppd_anychange[DEVNUM];
  hipStream_t streams[DEVNUM];
  
  //allocate device memory
  for(devIdx = 0; devIdx < devNum; devIdx++){
    hipSetDevice(devIdx);
    err = hipMalloc((void **)&ppd_vboxflat[devIdx], flatbytes);
    if(err != hipSuccess)
      printf("ppd_vboxflat malloc error\n");
    err = hipMalloc((void **)&ppd_anychange[devIdx], sizeof(int) * blkNum);
    if(err != hipSuccess)
      printf( "ppd_anychange malloc error\n");
    for(i=0; i<STARTMAX; i++){
      err = hipMalloc((void **)&pppd_ttboxflat[devIdx][i], flatbytes);
      if(err != hipSuccess)
        printf("pppd_ttboxflat malloc error\n");
    }
    err = hipMalloc((void **)&ppd_ttbflatMergeCache[devIdx], flatbytes*(devNum-1));
    if(err != hipSuccess)
      printf( "ppd_ttbflatMergeCache malloc error\n");
  }

  //lock up host memory for async transfer
  hipHostRegister(fs, sizeof(fs), hipHostRegisterDefault);
  hipHostRegister(start, sizeof(start), hipHostRegisterDefault);
  hipHostRegister(vbox.box.flat, flatbytes, hipHostRegisterDefault);
  for(i=0; i<STARTMAX; i++)
    hipHostRegister(ttboxes[i].flat, flatbytes, hipHostRegisterDefault);
  
  //async copy memory from host to device
  for(devIdx = 0; devIdx < devNum; devIdx++){
    hipSetDevice(devIdx);
    hipStreamCreate(&streams[devIdx]);
    
    //copy fs to device
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(dc_fs), fs, sizeof(fs), 0, hipMemcpyHostToDevice, streams[devIdx]);
    if(err != hipSuccess)
      printf("dc_fs copy error\n");
    printf("1\n");
    
    //copy velosity box to device
    err = hipMemcpyAsync(ppd_vboxflat[devIdx], vbox.box.flat, flatbytes, hipMemcpyHostToDevice, streams[devIdx]);
    if(err != hipSuccess)
      printf( "ppd_vboxflat copy error\n" );
    printf( "2\n" );
    
    //copy travel time boxes to device
    for(i=0; i<STARTMAX; i++){
      err = hipMemcpyAsync(pppd_ttboxflat[devIdx][i], ttboxes[i].flat, flatbytes, hipMemcpyHostToDevice, streams[devIdx]);
      if(err != hipSuccess)
        printf( "pppd_ttboxflat %d copy error\n", i );
    }
    printf("3\n");
  }
  hipStreamSynchronize(0);
  
  //run algorithm
  double tSweep = 0.0, tChangeTrans = 0.0, tSum = 0.0, tMerge = 0.0, tBoxTrans = 0.0, tTotal = 0.0;
  for(i=0; i<numstart; i++){
    int sweepNum = 0, changeSum = 1;
    while (changeSum) {//run until no changes
      changeSum = 0;
      sweepNum++;
      
      //run splited forward stars on different devices
      reset_and_start_timer();
      for(devIdx=0; devIdx<devNum; devIdx++){
        hipSetDevice(devIdx);
        err = hipMemset(ppd_anychange[devIdx], 0, sizeof(int) * blkNum);
        if(err != hipSuccess)
          printf( "ppd_anychange memset error\n");
        
        cudaWorker<<<gridDim,blockDim>>>(
          nx, ny, nz, 
          starSplit[devIdx], starSplit[devIdx+1]-1, //Note: change the range to the original starsize only reduce 5ms time.
          dc_fs, 
          ppd_vboxflat[devIdx], 
          pppd_ttboxflat[devIdx][i],
          ppd_anychange[devIdx]
        );
      }
      hipStreamSynchronize(0); //sync all devices
      tSweep = get_elapsed_msec();
      
      if(err != hipSuccess) //check error
        printf("  hipGetLastError() returned %d: %s\n", err, hipGetErrorString(err));
      
      //pull back and check changes
      reset_and_start_timer();
      for(devIdx=0; devIdx<devNum; devIdx++){
        hipSetDevice(devIdx);
        err = hipMemcpyAsync(anychange[devIdx], ppd_anychange[devIdx], sizeof(int) * blkNum, hipMemcpyDeviceToHost, streams[devIdx]);
        if(err != hipSuccess)
          printf("anychange copy error: %d\n", err);
      }
      hipStreamSynchronize(0); //sync all devices
      tChangeTrans = get_elapsed_msec();
      
      reset_and_start_timer();
      for(devIdx=0; devIdx<devNum; devIdx++)
        for(j = 0; j < blkNum; j++)
          changeSum += anychange[devIdx][j];
      tSum = get_elapsed_msec();
      
      //sync travel time from all devices and merge them
      reset_and_start_timer();
      printf("nCells = %d\n", nCells);
      float *pCacheBasePtr;
      int buffered, devFrom;
      for(devIdx=0; devIdx<devNum; devIdx++){
        hipSetDevice(devIdx);
        buffered = 0;
        for(devFrom=0; devFrom<devNum; devFrom++){
          if(devFrom != devIdx){
            pCacheBasePtr = ppd_ttbflatMergeCache[devIdx] + nCells*buffered;
            hipMemcpyPeerAsync(pCacheBasePtr, devIdx, pppd_ttboxflat[devFrom][i], devFrom, flatbytes, streams[devIdx]);
            buffered++;
          }
        }
      }    
      hipStreamSynchronize(0); //sync all devices
      tBoxTrans = get_elapsed_msec();
      reset_and_start_timer();
      
      for(devIdx=0; devIdx<devNum; devIdx++){
        hipSetDevice(devIdx);
        MergeBoxes<<<gridDim,blockDim,0,streams[devIdx]>>>(
          pppd_ttboxflat[devIdx][i], 
          ppd_ttbflatMergeCache[devIdx], 
          devNum, 
          nCells
        );
      }
      hipStreamSynchronize(0);
      tMerge = get_elapsed_msec();
      
      //output statistics
      tTotal = tSweep + tChangeTrans + tSum + tBoxTrans + tMerge;
      printf(" start point: %d, sweep %d: %d changes, sweep %g, change trans %g\n\
sum %g, box trans %g, merg %g, total %g\n", 
        i, sweepNum, changeSum, tSweep, tChangeTrans, tSum, tBoxTrans, tMerge, tTotal);
    }

    devIdx = 0;
    hipSetDevice(devIdx);
    err = hipMemcpy(ttboxes[i].flat, pppd_ttboxflat[devIdx][i], flatbytes, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
      printf( "pppd_ttboxflat %d copy error\n", i );
  }
	printf("6\n");
  
  for(devIdx=0; devIdx<devNum; devIdx++){
    hipFree(ppd_vboxflat[devIdx]);
    hipFree(ppd_anychange[devIdx]);
    for(i=0; i<STARTMAX; i++)
      err = hipFree(pppd_ttboxflat[devIdx][i]);
    hipFree(ppd_ttbflatMergeCache[devIdx]);
    hipStreamDestroy(streams[devIdx]);
  }
  
}

__global__ 
void cudaWorker(
    int d_nx, int d_ny, int d_nz,
    int d_starstart, int d_starend,
    struct FS *pd_fs,
    float *pd_vboxflat,
    float *pd_ttboxflat,
    int *pd_anychange
)
{
  int d_blktid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y;
  int d_blkid = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.z * gridDim.y;
	// int d_glbtid = d_blkid * (blockDim.x * blockDim.y * blockDim.z)
								// + (threadIdx.z * (blockDim.x * blockDim.y))
								// + (threadIdx.y * blockDim.x) + threadIdx.x;
  const int blkSize = BLOCKX*BLOCKY*BLOCKZ;
	__shared__ int blkChange[blkSize];
  
	blkChange[d_blktid] = sweepXYZ(
    d_nx, d_ny, d_nz,
    d_starstart, d_starend, 
    dc_fs, 
    pd_vboxflat,
    pd_ttboxflat
  );
  __syncthreads();

  //reduction
  for (int s=blkSize/2; s>0; s>>=1) {
    if (d_blktid < s)
      blkChange[d_blktid] += blkChange[d_blktid + s];
    __syncthreads();
  }
  // write result for this block to global mem
  if (d_blktid == 0) pd_anychange[d_blkid] = blkChange[0];
}

__device__ int 
sweepXYZ(
    int nx, int ny, int nz, 
    int starstart, int starstop,
    struct FS *fs,
    float *vboxflat,
    float *ttboxflat
) 
{
  int	i, j, k, l, oi, oj, ok, iIdx, oIdx;
  float	delay = 0.0, tt = 0.0, tto = 0.0, ttd = 0.0, ttod = 0.0;
  int sx = nz * ny;
  int d_blktid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y;
  __shared__ int change;
  if(d_blktid == 0)
    change = 0;
  __syncthreads();
  
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	k = blockIdx.z * blockDim.z + threadIdx.z;
	
	if(i >= nx || j >= ny || k >= nz)
		return 0;
	
	for (l=starstart; l<starstop; l++) {
		/* find point in forward star based on offsets */
		oi = i+fs[l].i; oj = j+fs[l].j; ok = k+fs[l].k;
		/* if (oi,oj,ok) is outside the boundaries, then skip */
		if ((oi < 0) || (oi > nx-1)
				|| (oj < 0) || (oj > ny-1)
				|| (ok < 0) || (ok > nz-1)) {
			continue;
		}
    
		//pre-compute all the needed values
    iIdx = k+nz*j+i*sx; oIdx = ok+nz*oj+oi*sx;
		delay = fs[l].d * (vboxflat[iIdx] + vboxflat[oIdx]) / 2.0;
		tt = ttboxflat[iIdx];
		tto = ttboxflat[oIdx];
    ttd = tt + delay;
    ttod = tto + delay;
    
    //if the difference between two values is greater than delay
    //do value switches using pre-calculated values.
    if(ttd < tto || ttod < tt){
      ttboxflat[iIdx] = fminf(tt, ttod);
      ttboxflat[oIdx] = fminf(tto, ttd);
      if(change == 0)
        change = 1;
    }
  }
  return(change);

} /* end sweepXYZ */ 

__global__ void 
MergeBoxes(
    float *pd_ttboxflat, 
    float *pd_ttbflatMergeCache, 
    int devNum, 
    int nCells)
{
  int d_blkid = blockIdx.z + blockIdx.y * gridDim.z + blockIdx.x * gridDim.z * gridDim.y;
	int d_glbtid = d_blkid * blockDim.x * blockDim.y * blockDim.z
								+ (threadIdx.x * blockDim.z * blockDim.y)
								+ (threadIdx.y * blockDim.z) + threadIdx.z;
  float res = INFINITY;
  res = fminf(res, pd_ttboxflat[d_glbtid]);
  int devFrom;
  for(devFrom=0; devFrom<devNum-1; devFrom++)
    if(d_glbtid < nCells)
      res = fminf(res, pd_ttbflatMergeCache[devFrom*nCells + d_glbtid]);
  pd_ttboxflat[d_glbtid] = res;
}